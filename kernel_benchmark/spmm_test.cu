#include "hip/hip_runtime.h"
/***************************************************************************
 * Copyright 2023 The HeteroSparse Authors. All rights reserved.
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ***************************************************************************/

// #define TRIM_B
// #define BREAKDOWN
#define SPARSE_LOAD_A

//#define USE_CUBLAS
#define USE_FLASH_LLM
// #define USE_SPUTNIK
// #define USE_CUSPARSE
//#define USE_SPARTA
//#define USE_CUSPARSELT

#include "./spmm_test_utils.h"
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include "FSP_Computation.h"
#include "FSP_Batch8.h"
#include <cstdlib>


#ifdef USE_FLASH_LLM
#include "SpMM_API.cuh"
#include "spmm_decompose.h"
#endif

// #ifdef USE_SPUTNIK
#include "./sputnik_utils.h"
#include "sputnik/sputnik.h"
// #endif

#ifdef USE_SPARTA
#include "sparTA.h"
#endif

#ifdef USE_CUSPARSELT
#include "CuSparseLt.h"
#endif
//

// ITERATION wrongly used in SPMM

int main(int argc, char** argv)
{
    if (argc != 7) {
        printf("Wrong Inputs! Correct input format: ./spmm_test M K N Sparsity A1_SplitK A2_SplitK\n");
        return;
    }
    int M_GLOBAL                    = atoi(argv[1]);
    int K_GLOBAL                    = atoi(argv[2]);
    int N_GLOBAL                    = atoi(argv[3]);
    int MATRIX_A_PRUNING_PERCENTAGE = atoi(argv[4]);
    int A1_SPLIT_K                     = atoi(argv[5]);
    int A2_SPLIT_K                     = atoi(argv[6]);

    if (N_GLOBAL > 8 && A2_SPLIT_K != 1) {
        printf("N_GLOBAL > 8, A2_SPLIT_K must be 1\n");
        return 0;
    }
    //
    // printf("M: %d N: %d K: %d\n", M_GLOBAL, N_GLOBAL, K_GLOBAL);
    //
    hipblasStatus_t cublas_status;
    // hipsparseStatus_t  cusparse_status;
    // hipError_t       cuda_error;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Host memory
    half* A_h            = NULL;  // row major
    half* B_h            = NULL;  // col major
    half* B_Transposed_h = NULL;  // row major
    // Device memory
    half* A            = NULL;
    half* B            = NULL;
    half* B_Transposed = NULL;
    //
    A_h            = (half*)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    B_h            = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
    B_Transposed_h = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
    if (A_h == NULL || B_h == NULL || B_Transposed_h == NULL) {
        printf("Error in CPU Malloc!\n");
        exit(-1);
    }
    hipMalloc(reinterpret_cast<void**>(&A), sizeof(half) * M_GLOBAL * K_GLOBAL);
    hipMalloc(reinterpret_cast<void**>(&B), sizeof(half) * N_GLOBAL * K_GLOBAL);
    hipMalloc(reinterpret_cast<void**>(&B_Transposed), sizeof(half) * N_GLOBAL * K_GLOBAL);
    checkLastCudaError(__LINE__);
    if (A == NULL || B == NULL || B_Transposed == NULL) {
        printf("Error in hipMalloc!\n");
        exit(-1);
    }
    //
    init_host_matrices(A_h, B_h, M_GLOBAL, K_GLOBAL, N_GLOBAL, MATRIX_A_PRUNING_PERCENTAGE);
    // init_host_structure_sparsity(A_h, B_h, M_GLOBAL, K_GLOBAL, N_GLOBAL, MATRIX_A_PRUNING_PERCENTAGE);
    // half *debug_A1, *debug_A2;
    // debug_A1 = (half *) malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    // debug_A2 = (half *) malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    // printf("decompose\n");
    // decompose(A_h, debug_A1, debug_A2, M_GLOBAL * K_GLOBAL);
    // //use A2 part to check
    // memcpy(A_h, debug_A1, sizeof(half) * M_GLOBAL * K_GLOBAL);
//    A_h[143*K_GLOBAL + 62] = 0.0;

//    int row_idx = 62;

//    for(int i = 0; i < N_GLOBAL; i++)
//    {
//        printf("%f ", __half2float(B_h[row_idx + i * K_GLOBAL]));
//    }

//    getchar();

//    InitUnStructureSparseMatrix(A2, M_GLOBAL, K_GLOBAL, NULL, NULL, NULL);
//    return 0;

//    init_host_structure_sparsity(A_h, B_h, M_GLOBAL, K_GLOBAL, N_GLOBAL, MATRIX_A_PRUNING_PERCENTAGE);

    for (int i = 0; i < K_GLOBAL; i++)
        for (int j = 0; j < N_GLOBAL; j++) {
            B_Transposed_h[i * N_GLOBAL + j] = B_h[i + j * K_GLOBAL];
            // printf("%f ", __half2float(B_Transposed_h[i * N_GLOBAL + j]));
        }
    //
    // printf("Preparing dense data for GPU...\n");
    hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    hipMemcpy(B_Transposed, B_Transposed_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
    //#ifdef USE_CUBLAS
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuBlas...\n");
    half* D_cublas = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_cublas), sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_cublas == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_cublas, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);

    // Tensor core not enabled
    hipblasSetMathMode(handle, HIPBLAS_PEDANTIC_MATH);
    hipDeviceSynchronize();
    int              m = M_GLOBAL, n = N_GLOBAL, k = K_GLOBAL;
    const float      alpha     = 1.0;
    const float      beta      = 0.0;
    hipblasGemmAlgo_t CuBlasALG = static_cast<hipblasGemmAlgo_t>(0);
    for (int i = 0; i < WARM_UP_ITERATION; i++) {
        cublas_status = hipblasGemmEx(handle,
                                     HIPBLAS_OP_T,
                                     HIPBLAS_OP_N,
                                     m,
                                     n,
                                     k,
                                     &alpha,
                                     A,
                                     HIP_R_16F,
                                     k,
                                     B,
                                     HIP_R_16F,
                                     k,
                                     &beta,
                                     D_cublas,
                                     HIP_R_16F,
                                     m,
                                     HIP_R_32F,
                                     CuBlasALG);
        checkCublasError(cublas_status, __LINE__);
    }
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        hipblasGemmEx(handle,
                     HIPBLAS_OP_T,
                     HIPBLAS_OP_N,
                     m,
                     n,
                     k,
                     &alpha,
                     A,
                     HIP_R_16F,
                     k,
                     B,
                     HIP_R_16F,
                     k,
                     &beta,
                     D_cublas,
                     HIP_R_16F,
                     m,
                     HIP_R_32F,
                     CuBlasALG);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //
    float milliseconds_cublas = 0;
    hipEventElapsedTime(&milliseconds_cublas, start, stop);
    milliseconds_cublas = milliseconds_cublas / BENCHMARK_ITERATION;
    float tflops_cublas =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_cublas / 1000.))
        / 1e12;
    // Tensor core enabled
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
    hipDeviceSynchronize();
    for (int i = 0; i < WARM_UP_ITERATION; i++) {
        cublas_status = hipblasGemmEx(handle,
                                     HIPBLAS_OP_T,
                                     HIPBLAS_OP_N,
                                     m,
                                     n,
                                     k,
                                     &alpha,
                                     A,
                                     HIP_R_16F,
                                     k,
                                     B,
                                     HIP_R_16F,
                                     k,
                                     &beta,
                                     D_cublas,
                                     HIP_R_16F,
                                     m,
                                     HIP_R_32F,
                                     CuBlasALG);
        checkCublasError(cublas_status, __LINE__);
    }
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        hipblasGemmEx(handle,
                     HIPBLAS_OP_T,
                     HIPBLAS_OP_N,
                     m,
                     n,
                     k,
                     &alpha,
                     A,
                     HIP_R_16F,
                     k,
                     B,
                     HIP_R_16F,
                     k,
                     &beta,
                     D_cublas,
                     HIP_R_16F,
                     m,
                     HIP_R_32F,
                     CuBlasALG);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //
    float milliseconds_cublas_tc = 0;
    hipEventElapsedTime(&milliseconds_cublas_tc, start, stop);
    milliseconds_cublas_tc = milliseconds_cublas_tc / BENCHMARK_ITERATION;
    float tflops_cublas_tc = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2)
                                                 / (milliseconds_cublas_tc / 1000.))
                             / 1e12;
    half* D_cublas_h = NULL;  // col major
    D_cublas_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_cublas_h == NULL) {
        printf("Error in spmm_test.cu: line %d CPU Malloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemcpy(D_cublas_h, D_cublas, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_cublas);
    /////////////////////////////////////////////////////////////////////////////////////////////////
//#endif
#ifdef USE_FLASH_LLM
    /////////////////////////////////////////////////////////////////////////////////////////////////
    half *A1, *A2;
    A1 = (half*)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    A2 = (half*)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    printf("decompose\n");
    decompose(A_h, A1, A2, M_GLOBAL * K_GLOBAL);
    // memset(A1, 0, sizeof(half) * M_GLOBAL * K_GLOBAL);
    // memcpy(A2, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL);

    half* D_SpMM2 = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_SpMM2), sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_SpMM2 == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_SpMM2, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    half* NZWeights_CPU   = NULL;
    int*      TileOffsets_CPU = NULL;
//    int NumOffsets = InitSparseMatrixA_API_NoReorder_v1(A_h, M_GLOBAL, N_GLOBAL, K_GLOBAL, &NZWeights_CPU, &TileOffsets_CPU);
    uint64_t* Ind_data_CPU = NULL; 
    uint64_t *Ind_data_GPU = NULL;
#ifdef SPARSE_LOAD_A
    int NumOffsets = InitSparseMatrixA1_Vec_Sparse_Load(A1, M_GLOBAL, K_GLOBAL, &NZWeights_CPU, &Ind_data_CPU, &TileOffsets_CPU);
#else
    int NumOffsets = InitSparseMatrixA_API_NoReorder_v4(A1, M_GLOBAL, N_GLOBAL, K_GLOBAL, &NZWeights_CPU, &TileOffsets_CPU);
#endif
    int NNZ        = TileOffsets_CPU[NumOffsets - 1] * 8;  // VectorSize = 4

    // bangtian: add meta data for using sparse tensor core
    int *h_meta, *d_meta;
    h_meta = (int *)malloc((M_GLOBAL/16) * (K_GLOBAL/16) * 8* sizeof(int));
    hipMalloc((void **)&d_meta, (M_GLOBAL/16) * (K_GLOBAL/16) * 8 * sizeof(int));
    inspect_metadata(A1, h_meta, M_GLOBAL, K_GLOBAL);
    hipMemcpy(d_meta, h_meta, (M_GLOBAL/16) * (K_GLOBAL/16) * 8*sizeof(int), hipMemcpyHostToDevice);

//

    // printf("NumOffsets: %d, NNZ: %d\n", NumOffsets, NNZ);
    //
    half *NZWeights_GPU   = NULL;
    int *TileOffsets_GPU = NULL;
    hipMalloc(&TileOffsets_GPU, sizeof(int) * NumOffsets);
    if (NNZ == 0)
        NNZ = 1;  // For 100% sparsity, NNZ = 0, malloc will return NULL
    hipMalloc(&NZWeights_GPU, sizeof(uint32_t) * NNZ);
    hipMalloc(reinterpret_cast<void**>(&NZWeights_GPU), sizeof(half) * NNZ);

#ifdef SPARSE_LOAD_A
    hipMalloc(reinterpret_cast<void**>(&Ind_data_GPU), sizeof(uint64_t) * (NNZ/8));
    hipMemcpy(Ind_data_GPU, Ind_data_CPU, sizeof(uint64_t) * (NNZ/8), hipMemcpyHostToDevice);
    free(Ind_data_CPU);
#endif

    if (TileOffsets_GPU == NULL || NZWeights_GPU == NULL) {
        printf("Error in malloc memory from device memory!\n");
        exit(-1);
    }
//    hipMemcpy(NZWeights_GPU, NZWeights_CPU, sizeof(uint32_t) * NNZ, hipMemcpyHostToDevice);
    hipMemcpy(NZWeights_GPU, NZWeights_CPU, sizeof(half) * NNZ, hipMemcpyHostToDevice);
    hipMemcpy(TileOffsets_GPU, TileOffsets_CPU, sizeof(int) * NumOffsets, hipMemcpyHostToDevice);
    
    free(TileOffsets_CPU);
    free(NZWeights_CPU);
    // printf("Done! Compressed A matrix for GPU kernel: MM_Sparse_TC.\n");
    //
    //Call Sputnik for A2 part
    float* A2_float_h = (float*)malloc(sizeof(float) * m * k);
    for (int i = 0; i < m * k; i++)
        A2_float_h[i] = __half2float(A2[i]);

    half *h_A2_vals, *d_A2_vals;
    uint16_t *h_A2_idx, *d_A2_idx;
    int *h_A2_row_ptr, *d_A2_row_ptr;
    int *h_row_length, *d_row_length;
    // below two are for sputnik-sf version
    uint8_t *h_A2_tile_length, *d_A2_tile_length;
    int *h_A2_tile_ptr, *d_A2_tile_ptr;
//     int size = A2_inspection_matrix(A2, M_GLOBAL, K_GLOBAL, &h_A2_vals, &h_A2_idx, &h_A2_row_ptr);
    int size;
    if (N_GLOBAL == 8)
        size = A2_inspection_matrix_batch8(A2, M_GLOBAL, K_GLOBAL, A2_SPLIT_K, &h_A2_vals, &h_A2_idx, &h_A2_row_ptr);
    else
        size = A2_inspection_matrix_sputnik_spadding(A2, M_GLOBAL, K_GLOBAL, &h_A2_vals, &h_A2_idx, &h_A2_row_ptr);
    
//    int size = A2_inspection_matrix_sputnik_spadding_sf(A2, M_GLOBAL, K_GLOBAL, &h_A2_vals, &h_A2_idx,
//                                                        &h_A2_tile_length, &h_A2_tile_ptr, &h_A2_row_ptr);


//    int size = A2_inspection_matrix_sputnik(A2, M_GLOBAL, K_GLOBAL, &h_A2_vals, &h_A2_idx, &h_A2_row_ptr, &h_row_length);
//    int size = A2_inspection_matrix_kcut_v1(A2, M_GLOBAL, K_GLOBAL, SPLIT_K, &h_A2_vals, &h_A2_idx, &h_A2_row_ptr);
//    return 0;
    hipMalloc((void **)&d_A2_vals, sizeof(half) * size);
    hipMalloc((void **)&d_A2_idx, sizeof(uint16_t) * size);
//    hipMalloc((void **)&d_A2_row_ptr, sizeof(int) * ((M_GLOBAL/32)*SPLIT_K + 1));
//    hipMalloc((void **)&d_A2_row_ptr, sizeof(int) * ((M_GLOBAL/32) + 1));
//    hipMalloc((void **)&d_A2_row_ptr, sizeof(int) * ((M_GLOBAL/4) + 1));
    hipMalloc((void **)&d_A2_row_ptr, sizeof(int) * ((M_GLOBAL) + 1));
//    hipMalloc((void **)&d_row_length, sizeof(int) * M_GLOBAL);

#ifdef TRIM_B
    // Compress fine-grained column indices
    int row_per_block = WARP_SIZE / (N_GLOBAL / HALF_PER_128B);
    for (int i = 0; i < size; i++) {
        int prev_col = h_A2_idx[i];

        int base_idx = prev_col - (prev_col % (row_per_block * 2));
        int remainder_idx = (prev_col % (row_per_block * 2));

        h_A2_idx[i] = base_idx + (remainder_idx - (2 - (remainder_idx % 2))) / 2;
    }
#endif

    hipMemcpy(d_A2_vals, h_A2_vals, sizeof(half) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_A2_idx, h_A2_idx, sizeof(uint16_t) * size, hipMemcpyHostToDevice);
//    hipMemcpy(d_A2_row_ptr, h_A2_row_ptr, sizeof(int) * ((M_GLOBAL/32)*SPLIT_K + 1), hipMemcpyHostToDevice);
//    hipMemcpy(d_A2_row_ptr, h_A2_row_ptr, sizeof(int) * ((M_GLOBAL/4) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_A2_row_ptr, h_A2_row_ptr, sizeof(int) * ((M_GLOBAL) + 1), hipMemcpyHostToDevice);
//    hipMemcpy(d_row_length, h_row_length, sizeof(int) * M_GLOBAL, hipMemcpyHostToDevice);


    int *row_indices = (int *)malloc(sizeof(int) * M_GLOBAL);
    for(int i=0; i < M_GLOBAL; i++)
    {
        row_indices[i] = i;
    }

    {
        // Create our unsorted row indices.
        std::vector<int> swizzle_staging(M_GLOBAL);
//        std::iota(swizzle_staging.begin(), swizzle_staging.end(), 0);
        for (int i=0; i < M_GLOBAL; i++)
            swizzle_staging[i] = i;
        // Argsort the row indices based on their length.
        std::sort(swizzle_staging.begin(), swizzle_staging.end(), [&h_A2_row_ptr](int idx_a, int idx_b) {
            int length_a = h_A2_row_ptr[idx_a + 1] - h_A2_row_ptr[idx_a];
            int length_b = h_A2_row_ptr[idx_b + 1] - h_A2_row_ptr[idx_b];
            return length_a > length_b;
        });

        // Copy the ordered row indices to the output.
        std::memcpy(row_indices, swizzle_staging.data(), sizeof(int) * M_GLOBAL);
    }

    int *d_row_indices;
    hipMalloc((void **)&d_row_indices, sizeof(int) * M_GLOBAL);
    hipMemcpy(d_row_indices, row_indices, sizeof(int) * M_GLOBAL, hipMemcpyHostToDevice);


//    hipMalloc((void **)&d_A2_tile_length, sizeof(uint8_t) * (M_GLOBAL/4));
//    hipMemcpy(d_A2_tile_length, h_A2_tile_length, sizeof(uint8_t) * (M_GLOBAL/4), hipMemcpyHostToDevice);
//    hipMalloc((void **)&d_A2_tile_ptr, sizeof(int) * (M_GLOBAL/4));
//    hipMemcpy(d_A2_tile_ptr, h_A2_tile_ptr, sizeof(int) * (M_GLOBAL/4), hipMemcpyHostToDevice);

    checkLastCudaError(__LINE__);
//    return 0;
//
//    sputnik_utils::SparseMatrix            sparse_matrix_A2(m, k, A2_float_h, sputnik_utils::SORTED, 4);
//    sputnik_utils::CudaSparseMatrix<half2> sparse_matrix_A2_gpu(sparse_matrix_A2);

    printf("Launching HeteroSparse without Ahead of Time Sparse Data Reordering...\n");
    
    half *Reduction_Workspace = NULL;
    hipMalloc(reinterpret_cast<void**>(&Reduction_Workspace), sizeof(half) * M_GLOBAL * N_GLOBAL * (A1_SPLIT_K + A2_SPLIT_K));
    hipMemset(Reduction_Workspace, 0, sizeof(half) * M_GLOBAL * N_GLOBAL * (A1_SPLIT_K + A2_SPLIT_K));
    if (Reduction_Workspace == NULL) {
        printf("Error in hipMalloc\n");
        exit(-1);
    }

    // Reduction kernel configuration
    dim3 grid2(M_GLOBAL / 32, N_GLOBAL / 8, 1);
    dim3 block2(32, 1, 1);

    // Initialize separate streams for A1 and A2
    hipStream_t stream_A1, stream_A2, stream_reduction;
    hipStreamCreate(&stream_A1);
    hipStreamCreate(&stream_A2);
    hipStreamCreate(&stream_reduction);

    hipProfilerStart();
    for (int i = 0; i < WARM_UP_ITERATION; i++){
        SpMM_SplitK_API(0,
                       A,
                       reinterpret_cast<uint4 *>(NZWeights_GPU),
                       Ind_data_GPU,
                       d_meta,
                       TileOffsets_GPU,
                       B_Transposed,
                       D_SpMM2,
                       M_GLOBAL,
                       N_GLOBAL,
                       K_GLOBAL,
                       Reduction_Workspace,
                       A1_SPLIT_K);
        if (N_GLOBAL == 8)
            CUDA_CALL(FSP_Batch8(0,
                                M_GLOBAL,
                                N_GLOBAL,
                                K_GLOBAL,
                                A2_SPLIT_K,
                                d_A2_vals,
                                d_A2_idx,
                                d_A2_row_ptr,
                                // d_row_indices,
                                B_Transposed,
                                D_SpMM2,
                                Reduction_Workspace + M_GLOBAL * N_GLOBAL * A1_SPLIT_K
                                ));
        else
            CUDA_CALL(FSP_Computation_reorder(M_GLOBAL,
                                            N_GLOBAL,
                                            K_GLOBAL,
                                            d_A2_vals,
                                            d_A2_idx,
                                            d_A2_row_ptr,
                                            d_row_indices,
                                            B_Transposed,
                                            Reduction_Workspace + M_GLOBAL * N_GLOBAL * A1_SPLIT_K));
        // hipDeviceSynchronize();
        Reduction<<<grid2, block2, 0, 0>>>(D_SpMM2, Reduction_Workspace, M_GLOBAL, N_GLOBAL, K_GLOBAL, (A1_SPLIT_K + A2_SPLIT_K));
        // hipDeviceSynchronize();
    }

    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++) {
        SpMM_SplitK_API(0,
                       A,
                       reinterpret_cast<uint4 *>(NZWeights_GPU),
                       Ind_data_GPU,
                       d_meta,
                       TileOffsets_GPU,
                       B_Transposed,
                       D_SpMM2,
                       M_GLOBAL,
                       N_GLOBAL,
                       K_GLOBAL,
                       Reduction_Workspace,
                       A1_SPLIT_K);
        if (N_GLOBAL == 8)
            CUDA_CALL(FSP_Batch8(0,
                                M_GLOBAL,
                                N_GLOBAL,
                                K_GLOBAL,
                                A2_SPLIT_K,
                                d_A2_vals,
                                d_A2_idx,
                                d_A2_row_ptr,
                                // d_row_indices,
                                B_Transposed,
                                D_SpMM2,
                                Reduction_Workspace + M_GLOBAL * N_GLOBAL * A1_SPLIT_K
                                ));
        else
            CUDA_CALL(FSP_Computation_reorder(M_GLOBAL,
                                            N_GLOBAL,
                                            K_GLOBAL,
                                            d_A2_vals,
                                            d_A2_idx,
                                            d_A2_row_ptr,
                                            d_row_indices,
                                            B_Transposed,
                                            Reduction_Workspace + M_GLOBAL * N_GLOBAL * A1_SPLIT_K));
        // hipDeviceSynchronize();
        Reduction<<<grid2, block2, 0, 0>>>(D_SpMM2, Reduction_Workspace, M_GLOBAL, N_GLOBAL, K_GLOBAL, (A1_SPLIT_K + A2_SPLIT_K));
        // hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    checkLastCudaError(__LINE__);
    hipProfilerStop();
    // Get the average of the times
    float milliseconds_SpMM2 = 0.0f;
    hipEventElapsedTime(&milliseconds_SpMM2, start, stop);
    milliseconds_SpMM2 = milliseconds_SpMM2 / BENCHMARK_ITERATION;
    float tflops_SpMM2 =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_SpMM2 / 1000.))
        / 1e12;
    half* D_SpMM_h2 = NULL;  // col major
    D_SpMM_h2       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipMemcpy(D_SpMM_h2, D_SpMM2, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major

    half* D_SpMM_h2_col_maj = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    for(int i = 0; i < M_GLOBAL; i++)
    {
        for(int j = 0; j < N_GLOBAL; j++)
        {
            D_SpMM_h2_col_maj[i + j * M_GLOBAL] = D_SpMM_h2[j + i * N_GLOBAL];
        }
    }


#ifdef BREAKDOWN
    // Breakdown Times
    for (int i = 0; i < WARM_UP_ITERATION; i++){
        SpMM_SplitK_API(0,
                       A,
                       reinterpret_cast<uint4 *>(NZWeights_GPU),
                    //    Ind_data_GPU,
                       d_meta,
                       TileOffsets_GPU,
                       B_Transposed,
                       D_SpMM2,
                       M_GLOBAL,
                       N_GLOBAL,
                       K_GLOBAL,
                       A1_Reduction_Workspace,
                       A1_SPLIT_K);
    }
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++) {
        SpMM_SplitK_API(0,
                       A,
                       reinterpret_cast<uint4 *>(NZWeights_GPU),
                    //    Ind_data_GPU,
                       d_meta,
                       TileOffsets_GPU,
                       B_Transposed,
                       D_SpMM2,
                       M_GLOBAL,
                       N_GLOBAL,
                       K_GLOBAL,
                       A1_Reduction_Workspace,
                       A1_SPLIT_K);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // Get the average of the times
    float milliseconds_SpMM2_A1 = 0.0f;
    hipEventElapsedTime(&milliseconds_SpMM2_A1, start, stop);
    milliseconds_SpMM2_A1 = milliseconds_SpMM2_A1 / BENCHMARK_ITERATION;
    float tflops_SpMM2_A1 =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_SpMM2_A1 / 1000.))
        / 1e12;

    for (int i = 0; i < WARM_UP_ITERATION; i++){
        CUDA_CALL(FSP_Batch8(M_GLOBAL,
                            N_GLOBAL,
                            K_GLOBAL,
                            A2_SPLIT_K,
                            d_A2_vals,
                            d_A2_idx,
                            d_A2_row_ptr,
                            // d_row_indices,
                            B_Transposed,
                            D_SpMM2,
                            A2_Reduction_Workspace
                            ));
    }
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++) {
        CUDA_CALL(FSP_Batch8(M_GLOBAL,
                            N_GLOBAL,
                            K_GLOBAL,
                            A2_SPLIT_K,
                            d_A2_vals,
                            d_A2_idx,
                            d_A2_row_ptr,
                            // d_row_indices,
                            B_Transposed,
                            D_SpMM2,
                            A2_Reduction_Workspace
                            ));
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // Get the average of the times
    float milliseconds_SpMM2_A2 = 0.0f;
    hipEventElapsedTime(&milliseconds_SpMM2_A2, start, stop);
    milliseconds_SpMM2_A2 = milliseconds_SpMM2_A2 / BENCHMARK_ITERATION;
    float tflops_SpMM2_A2 =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_SpMM2_A2 / 1000.))
        / 1e12;
#endif

    // std::cout.unsetf(std::ios::floatfield); std::cout << std::setprecision(4);
    // int stride = 2;
    // std::cout << "D_SpMM_h2:" << std::endl;
    // for (int i = 0; i < 10; i++) {
    //     for (int j = 0; j < std::min(64, N_GLOBAL) / stride; j++) {
    //         std::cout << __half2float(D_SpMM_h2_col_maj[i + j * stride * M_GLOBAL]) << "\t";
    //     }
    //     std::cout << std::endl;
    //     std::cout << std::endl;
    // }
    // std::cout << "D_cublas_h:" << std::endl;
    // for (int i = 0; i < 10; i++) {
    //     for (int j = 0; j < std::min(64, N_GLOBAL) / stride; j++) {
    //         std::cout << __half2float(D_cublas_h[i + j * stride * M_GLOBAL]) << "\t";
    //     }
    //     std::cout << std::endl;
    //     std::cout << std::endl;
    // }
    // std::cout.unsetf(std::ios::floatfield);

    // for (int i = 0; i < M_GLOBAL; i++) {
    //     for (int j = 0; j < N_GLOBAL; j += stride) {
    //         if (isnan(__half2float(D_SpMM_h2_col_maj[i + j * M_GLOBAL])) || fabs(__half2float(D_SpMM_h2_col_maj[i + j * M_GLOBAL]) - __half2float(D_cublas_h[i + j * M_GLOBAL])) > 0.9) {
    //             printf("i = %d, j = %d, D_SpMM_h2 = %f, D_cublas_h = %f\n", i, j, __half2float(D_SpMM_h2_col_maj[i + j * M_GLOBAL]), __half2float(D_cublas_h[i + j * M_GLOBAL]));
    //             getchar();
    //         }
    //     }
    // }

    free(D_SpMM_h2);
    hipFree(D_SpMM2);
    hipFree(NZWeights_GPU);
    hipFree(TileOffsets_GPU);
    hipFree(Reduction_Workspace);
    /////////////////////////////////////////////////////////////////////////////////////////////////
#endif
#ifdef USE_CUSPARSE
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuSparse_ColMajor...\n");
    half* D_CuSparse = NULL;
    hipMalloc(&D_CuSparse, N_GLOBAL * M_GLOBAL * sizeof(half));
    if (D_CuSparse == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_CuSparse, 0.0f, N_GLOBAL * M_GLOBAL * sizeof(half));
    //
    hipsparseHandle_t sp_handle = 0;
    hipsparseCreate(&sp_handle);
    hipsparseSetStream(sp_handle, 0);
    hipsparseSpMatDescr_t SpMatA;
    hipsparseDnMatDescr_t DnMatA, DnMatB, DnMatC;
    // Create Dense Matrix
    CHECK_CUSPARSE(hipsparseCreateDnMat(&DnMatA,
                                       M_GLOBAL,
                                       K_GLOBAL,
                                       K_GLOBAL,
                                       A,
                                       HIP_R_16F,
                                       HIPSPARSE_ORDER_ROW))  // Very critical!!! Weight Matrix must be row major,
                                                             // otherwise causing significant performance problems
    CHECK_CUSPARSE(hipsparseCreateDnMat(&DnMatB, K_GLOBAL, N_GLOBAL, K_GLOBAL, B, HIP_R_16F, HIPSPARSE_ORDER_COL))
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&DnMatC, M_GLOBAL, N_GLOBAL, M_GLOBAL, D_CuSparse, HIP_R_16F, HIPSPARSE_ORDER_COL))
    // Create Sparse Matrix in CSR format
    int* csrRowPtr;
    hipMalloc(&csrRowPtr, sizeof(int) * (M_GLOBAL + 1));
    CHECK_CUSPARSE(hipsparseCreateCsr(&SpMatA,
                                     M_GLOBAL,
                                     K_GLOBAL,
                                     0,
                                     csrRowPtr,
                                     NULL,
                                     NULL,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_16F))
    // execute Sparse to Dense conversion
    void*  Buffer     = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_bufferSize(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize))
    hipMalloc(&Buffer, bufferSize);
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_analysis(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer))
    //
    int64_t numRowTMP, numColTMP, NNZ_1;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(SpMatA, &numRowTMP, &numColTMP, &NNZ_1))
    //
    int*  csrColInd;
    half* csrVal;
    hipMalloc(&csrColInd, NNZ_1 * sizeof(int));
    hipMalloc(&csrVal, NNZ_1 * sizeof(half));
    //
    CHECK_CUSPARSE(hipsparseCsrSetPointers(SpMatA, csrRowPtr, csrColInd, csrVal))
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer))
    //
    hipsparseSpMMAlg_t CuSparse_Algorithm;
    CuSparse_Algorithm = HIPSPARSE_SPMM_ALG_DEFAULT;
    CuSparse_Algorithm =
        HIPSPARSE_SPMM_CSR_ALG1;  // csrmm_kernel faster: Provide the best performance with column-major layout
    // CuSparse_Algorithm = HIPSPARSE_SPMM_CSR_ALG2;    // csrmm_v2_kernel: Provide the best performance with row-major
    // layout!!! How about try row major of B&C? CuSparse_Algorithm = HIPSPARSE_SPMM_CSR_ALG3;
    //
    // printf("CuSparse Algorithm: %d \n", CuSparse_Algorithm);
    //
    const float alpha_float = 1.0;
    const float beta_float  = 0.0;
    //
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(sp_handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha_float,
                                           SpMatA,
                                           DnMatB,
                                           &beta_float,
                                           DnMatC,
                                           HIP_R_32F,
                                           CuSparse_Algorithm,
                                           &bufferSize))
    hipFree(Buffer);
    hipMalloc(&Buffer, bufferSize);
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA,
                                    DnMatB,
                                    &beta_float,
                                    DnMatC,
                                    HIP_R_32F,
                                    CuSparse_Algorithm,
                                    Buffer))
    hipEventRecord(start);
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA,
                                    DnMatB,
                                    &beta_float,
                                    DnMatC,
                                    HIP_R_32F,
                                    CuSparse_Algorithm,
                                    Buffer))
    hipEventRecord(stop);
    //
    float milliseconds_CuSparse_ColMajor = 0.0f;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_CuSparse_ColMajor, start, stop);
    milliseconds_CuSparse_ColMajor = milliseconds_CuSparse_ColMajor / CUSPARSE_ITERATION;
    float tflops_CuSparse_ColMajor = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2)
                                                         / (milliseconds_CuSparse_ColMajor / 1000.))
                                     / 1e12;
    //
    half* D_CuSparse_h;
    D_CuSparse_h = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_CuSparse_h == NULL) {
        printf("Error in spmm_test.cu: line %d CPU Malloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemcpy(D_CuSparse_h, D_CuSparse, N_GLOBAL * M_GLOBAL * sizeof(half), hipMemcpyDeviceToHost);
    hipFree(D_CuSparse);
    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);
    hipFree(Buffer);
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuSparse_RowMajor...\n");
    half* D_CuSparse_2;
    hipMalloc(&D_CuSparse_2, N_GLOBAL * M_GLOBAL * sizeof(half));
    if (D_CuSparse_2 == NULL) {
        printf("Error in Test_SpMM_v2.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_CuSparse_2, 0.0f, N_GLOBAL * M_GLOBAL * sizeof(half));
    //
    hipsparseHandle_t sp_handle_2 = 0;
    hipsparseCreate(&sp_handle_2);
    hipsparseSpMatDescr_t SpMatA_2;
    hipsparseDnMatDescr_t DnMatA_2, DnMatB_2, DnMatC_2;
    // Create Dense Matrix
    CHECK_CUSPARSE(hipsparseCreateDnMat(&DnMatA_2,
                                       M_GLOBAL,
                                       K_GLOBAL,
                                       K_GLOBAL,
                                       A,
                                       HIP_R_16F,
                                       HIPSPARSE_ORDER_ROW))  // Very critical!!! Weight Matrix must be row major,
                                                             // otherwise causing significant performance problems

    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&DnMatB_2, K_GLOBAL, N_GLOBAL, N_GLOBAL, B_Transposed, HIP_R_16F, HIPSPARSE_ORDER_ROW))
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&DnMatC_2, M_GLOBAL, N_GLOBAL, N_GLOBAL, D_CuSparse_2, HIP_R_16F, HIPSPARSE_ORDER_ROW))
    // Create Sparse Matrix in CSR format
    int* csrRowPtr_2;
    hipMalloc(&csrRowPtr_2, sizeof(int) * (M_GLOBAL + 1));
    CHECK_CUSPARSE(hipsparseCreateCsr(&SpMatA_2,
                                     M_GLOBAL,
                                     K_GLOBAL,
                                     0,
                                     csrRowPtr_2,
                                     NULL,
                                     NULL,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_16F))
    // execute Sparse to Dense conversion
    void*  Buffer_2     = NULL;
    size_t bufferSize_2 = 0;
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(
        sp_handle_2, DnMatA_2, SpMatA_2, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize_2))
    hipMalloc(&Buffer_2, bufferSize_2);
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_analysis(sp_handle_2, DnMatA_2, SpMatA_2, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer_2))
    //
    int64_t numRowTMP_2, numColTMP_2, NNZ_2;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(SpMatA_2, &numRowTMP_2, &numColTMP_2, &NNZ_2))
    //
    int*  csrColInd_2;
    half* csrVal_2;
    hipMalloc(&csrColInd_2, NNZ_2 * sizeof(int));
    hipMalloc(&csrVal_2, NNZ_2 * sizeof(half));
    //
    CHECK_CUSPARSE(hipsparseCsrSetPointers(SpMatA_2, csrRowPtr_2, csrColInd_2, csrVal_2))
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_convert(sp_handle_2, DnMatA_2, SpMatA_2, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer_2))
    //
    hipsparseSpMMAlg_t CuSparse_Algorithm_2;
    CuSparse_Algorithm_2 = HIPSPARSE_SPMM_ALG_DEFAULT;
    CuSparse_Algorithm_2 =
        HIPSPARSE_SPMM_CSR_ALG1;  // csrmm_kernel faster: Provide the best performance with column-major layout
    CuSparse_Algorithm_2 = HIPSPARSE_SPMM_CSR_ALG2;  // csrmm_v2_kernel: Provide the best performance with row-major
                                                    // layout!!! How about try row major of B&C?
    // CuSparse_Algorithm_2 = HIPSPARSE_SPMM_CSR_ALG3;
    // printf("CuSparse Algorithm: %d \n", CuSparse_Algorithm_2);
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(sp_handle_2,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha_float,
                                           SpMatA_2,
                                           DnMatB_2,
                                           &beta_float,
                                           DnMatC_2,
                                           HIP_R_32F,
                                           CuSparse_Algorithm_2,
                                           &bufferSize_2))
    hipFree(Buffer_2);
    hipMalloc(&Buffer_2, bufferSize_2);
    //
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle_2,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA_2,
                                    DnMatB_2,
                                    &beta_float,
                                    DnMatC_2,
                                    HIP_R_32F,
                                    CuSparse_Algorithm_2,
                                    Buffer_2))
    hipEventRecord(start);
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle_2,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA_2,
                                    DnMatB_2,
                                    &beta_float,
                                    DnMatC_2,
                                    HIP_R_32F,
                                    CuSparse_Algorithm_2,
                                    Buffer_2))
    hipEventRecord(stop);
    //
    float milliseconds_CuSparse_RowMajor;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_CuSparse_RowMajor, start, stop);
    milliseconds_CuSparse_RowMajor = milliseconds_CuSparse_RowMajor / CUSPARSE_ITERATION;
    float tflops_CuSparse_RowMajor = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2)
                                                         / (milliseconds_CuSparse_RowMajor / 1000.))
                                     / 1e12;
    // transpose result to col-major
    half* D_CuSparse_h_2_row_major;
    half* D_CuSparse_h_2;
    D_CuSparse_h_2           = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    D_CuSparse_h_2_row_major = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_CuSparse_h_2 == NULL || D_CuSparse_h_2_row_major == NULL) {
        printf("Error in spmm_test.cu: line %d CPU Malloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemcpy(D_CuSparse_h_2_row_major,
               D_CuSparse_2,
               N_GLOBAL * M_GLOBAL * sizeof(half),
               hipMemcpyDeviceToHost);  // row major
    for (int i = 0; i < N_GLOBAL; i++)
        for (int j = 0; j < M_GLOBAL; j++)
            D_CuSparse_h_2[i * M_GLOBAL + j] = D_CuSparse_h_2_row_major[i + j * N_GLOBAL];
    free(D_CuSparse_h_2_row_major);
    hipFree(D_CuSparse_2);
    hipFree(csrRowPtr_2);
    hipFree(csrColInd_2);
    hipFree(csrVal_2);
    hipFree(Buffer_2);
    /////////////////////////////////////////////////////////////////////////////////////////////////
#endif
#ifdef USE_SPUTNIK
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching Sputnik...\n");
    half* D_Sputnik = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_Sputnik), sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_Sputnik == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_Sputnik, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    //
    float* A_float_h = NULL;
    A_float_h        = (float*)malloc(sizeof(float) * M_GLOBAL * K_GLOBAL);
    for (int i = 0; i < M_GLOBAL * K_GLOBAL; i++)
        A_float_h[i] = __half2float(A_h[i]);
    sputnik_utils::SparseMatrix            sparse_matrix(M_GLOBAL, K_GLOBAL, A_float_h, sputnik_utils::IDENTITY, 4);
    sputnik_utils::CudaSparseMatrix<half2> sparse_matrix_gpu(sparse_matrix);
    for (int i = 0; i < WARM_UP_ITERATION; i++)
        CUDA_CALL(sputnik::CudaSpmm(M_GLOBAL,
                                    K_GLOBAL,
                                    N_GLOBAL,
                                    sparse_matrix_gpu.NumElementsWithPadding(),
                                    sparse_matrix_gpu.RowIndices(),
                                    sparse_matrix_gpu.Values(),
                                    sparse_matrix_gpu.RowOffsets(),
                                    sparse_matrix_gpu.ColumnIndices(),
                                    reinterpret_cast<half2*>(B_Transposed),
                                    reinterpret_cast<half2*>(D_Sputnik),
                                    0));
    
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++) {
        CUDA_CALL(sputnik::CudaSpmm(M_GLOBAL,
                                    K_GLOBAL,
                                    N_GLOBAL,
                                    sparse_matrix_gpu.NumElementsWithPadding(),
                                    sparse_matrix_gpu.RowIndices(),
                                    sparse_matrix_gpu.Values(),
                                    sparse_matrix_gpu.RowOffsets(),
                                    sparse_matrix_gpu.ColumnIndices(),
                                    reinterpret_cast<half2*>(B_Transposed),
                                    reinterpret_cast<half2*>(D_Sputnik),
                                    0));
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds_Sputnik = 0;
    hipEventElapsedTime(&milliseconds_Sputnik, start, stop);
    milliseconds_Sputnik = milliseconds_Sputnik / BENCHMARK_ITERATION;
    float tflops_Sputnik =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_Sputnik / 1000.))
        / 1e12;
    // transpose result to col-major
    half* D_Sputnik_h_row_major = NULL;
    half* D_Sputnik_h           = NULL;
    D_Sputnik_h_row_major       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    D_Sputnik_h                 = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipMemcpy(
        D_Sputnik_h_row_major, D_Sputnik, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_Sputnik);
    for (int i = 0; i < N_GLOBAL; i++)
        for (int j = 0; j < M_GLOBAL; j++)
            D_Sputnik_h[i * M_GLOBAL + j] = D_Sputnik_h_row_major[i + j * N_GLOBAL];
    free(D_Sputnik_h_row_major);
    /////////////////////////////////////////////////////////////////////////////////////////////////
#endif
#ifdef USE_SPARTA
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching sparTA...\n");
    //
    float milliseconds_sparTA  = 0;
    half* D_sparTA_h_row_major = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_sparTA_h_row_major == NULL) {
        printf("Error in spmm_test.cu: line %d Malloc falied\n", __LINE__);
        exit(-1);
    }
    sparTA(A_h, B_Transposed_h, D_sparTA_h_row_major, M_GLOBAL, N_GLOBAL, K_GLOBAL, &milliseconds_sparTA);
    float tflops_sparTA =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_sparTA / 1000.))
        / 1e12;
    // transpose result to col-major
    half* D_sparTA_h = NULL;
    D_sparTA_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    for (int i = 0; i < N_GLOBAL; i++)
        for (int j = 0; j < M_GLOBAL; j++)
            D_sparTA_h[i * M_GLOBAL + j] = D_sparTA_h_row_major[i + j * N_GLOBAL];
    free(D_sparTA_h_row_major);
    /////////////////////////////////////////////////////////////////////////////////////////////////
#endif
#ifdef USE_CUSPARSELT
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuSparseLt...\n");
    //
    float milliseconds_cusparselt  = 0;
    half* D_cusparselt_h_row_major = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_cusparselt_h_row_major == NULL) {
        printf("Error in spmm_test.cu: line %d Malloc falied\n", __LINE__);
        exit(-1);
    }
    CuSparseLt_matmul(A_h, B_Transposed_h, D_cusparselt_h_row_major, M_GLOBAL, N_GLOBAL, K_GLOBAL, &milliseconds_cusparselt);
    float tflops_cusparselt =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_cusparselt / 1000.))
        / 1e12;
    // transpose result to col-major
    half* D_cusparselt_h = NULL;
    D_cusparselt_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    for (int i = 0; i < N_GLOBAL; i++)
        for (int j = 0; j < M_GLOBAL; j++)
            D_cusparselt_h[i * M_GLOBAL + j] = D_cusparselt_h_row_major[i + j * N_GLOBAL];
    free(D_cusparselt_h_row_major);
    /////////////////////////////////////////////////////////////////////////////////////////////////
#endif
    // printf("Verifying correctness of the computations...\n");
    //
#ifdef USE_FLASH_LLM
//    double totalError_SpMM  = ComputeTotalError(D_cublas_h, D_SpMM_h, M_GLOBAL, N_GLOBAL);
    double totalError_SpMM2 = ComputeTotalError(D_cublas_h, D_SpMM_h2_col_maj, M_GLOBAL, N_GLOBAL);
    // PrintMismatch("MySpMM", 10, 0.5, D_cublas_h, D_SpMM_h, M_GLOBAL, N_GLOBAL);
    free(D_SpMM_h2_col_maj);
#endif
#ifdef USE_CUSPARSE
    double totalError_CuSparse   = Comput eTotalError(D_cublas_h, D_CuSparse_h, M_GLOBAL, N_GLOBAL);
    double totalError_CuSparse_2 = ComputeTotalError(D_cublas_h, D_CuSparse_h_2, M_GLOBAL, N_GLOBAL);
    // PrintMismatch("CuSparse",   10, 0.5, D_cublas_h, D_CuSparse_h, M_GLOBAL, N_GLOBAL);
    // PrintMismatch("CuSparse2",  10, 0.5, D_cublas_h, D_CuSparse_h_2, M_GLOBAL, N_GLOBAL);
    free(D_CuSparse_h);
    free(D_CuSparse_h_2);
#endif
#ifdef USE_SPUTNIK
    double totalError_Sputnik = ComputeTotalError(D_cublas_h, D_Sputnik_h, M_GLOBAL, N_GLOBAL);
    // PrintMismatch("Sputnik", 10, 0.5, D_cublas_h, D_Sputnik_h, M_GLOBAL, N_GLOBAL);
    free(D_Sputnik_h);
#endif
#ifdef USE_SPARTA
    double totalError_sparTA = ComputeTotalError(D_cublas_h, D_sparTA_h, M_GLOBAL, N_GLOBAL);
    // PrintMismatch("sparTA", 10, 0.5, D_cublas_h, D_sparTA_h, M_GLOBAL, N_GLOBAL);
    free(D_sparTA_h);
#endif
#ifdef USE_CUSPARSELT
    double totalError_cusparselt = ComputeTotalError(D_cublas_h, D_cusparselt_h, M_GLOBAL, N_GLOBAL);
    // PrintMismatch("CuSparseLt", 10, 0.5, D_cublas_h, D_cusparselt_h, M_GLOBAL, N_GLOBAL);
    free(D_cusparselt_h);
#endif
    printf("******************************************Problem Size******************************************\n");
    printf("M: %d N: %d K: %d Pruning Rate: %d A1_SplitK: %d A2_SplitK: %d\n",
           M_GLOBAL,
           N_GLOBAL,
           K_GLOBAL,
           MATRIX_A_PRUNING_PERCENTAGE,
           A1_SPLIT_K,
           A2_SPLIT_K);
// printf("******************************************Performance*******************************************\n");
#ifdef USE_CUSPARSE
    PrintPerformance("CuSparse_C", milliseconds_CuSparse_ColMajor, tflops_CuSparse_ColMajor, totalError_CuSparse);
    PrintPerformance("CuSparse_R", milliseconds_CuSparse_RowMajor, tflops_CuSparse_RowMajor, totalError_CuSparse_2);
#endif
#ifdef USE_SPUTNIK
    PrintPerformance("Sputnik", milliseconds_Sputnik, tflops_Sputnik, totalError_Sputnik);
#endif
#ifdef USE_CUBLAS
    PrintPerformance("CuBlas_SIMT", milliseconds_cublas, tflops_cublas, 0.0);
    PrintPerformance("CuBlas_TC", milliseconds_cublas_tc, tflops_cublas_tc, 0.0);
#endif
#ifdef USE_SPARTA
    PrintPerformance("sparTA", milliseconds_sparTA, tflops_sparTA, totalError_sparTA);
#endif
#ifdef USE_CUSPARSELT
    PrintPerformance("CuSparseLt", milliseconds_cusparselt, tflops_cusparselt, totalError_cusparselt);
#endif
#ifdef USE_FLASH_LLM
    PrintPerformance("HeteroSparse_v1", milliseconds_SpMM2, tflops_SpMM2, totalError_SpMM2);
#ifdef BREAKDOWN
    PrintPerformance("HeteroSparse_v1_A1", milliseconds_SpMM2_A1, tflops_SpMM2_A1, NAN);
    PrintPerformance("HeteroSparse_v1_A2", milliseconds_SpMM2_A2, tflops_SpMM2_A2, NAN);
//    PrintPerformance("HeteroSparse_v2", milliseconds_SpMM, tflops_SpMM, totalError_SpMM);
#endif
#endif

    free(D_cublas_h);
    free(A_h);
    free(B_h);
    free(B_Transposed_h);
    hipFree(A);
    hipFree(B);
    hipFree(B_Transposed);
#ifdef USE_FLASH_LLM
    free(h_meta);
    hipFree(d_meta);
//    hipFree(A2_vals_gpu);
//    hipFree(A2_idx_gpu);
//    free(A1);
//    free(A2);
#endif
    return 0;
}
